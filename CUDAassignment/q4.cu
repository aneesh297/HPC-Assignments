#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define N 1024
int a[N][N],b[N][N],c[N][N];
using namespace std;

__global__ void addMatrix(int a[][N], int b[][N], int c[][N], int n){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if(row<n && col<n)
		c[row][col] = a[row][col] + b[row][col];
}

void random_int(int a[][N], int n)
{
   int i,j;
   for (i = 0; i < n; ++i)
   	for (j = 0; j < n; ++j)
    	a[i][j] = rand() % 101;

}

int main(void)
{
	
	int (*pA)[N], (*pB)[N], (*pC)[N];

	random_int(a,N);
	random_int(b,N);

	hipMalloc((void**)&pA, (N*N)*sizeof(int));
	hipMalloc((void**)&pB, (N*N)*sizeof(int));
	hipMalloc((void**)&pC, (N*N)*sizeof(int));

	hipMemcpy(pA, a, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pB, b, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pC, c, (N*N)*sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(64, 64);
    dim3 dimGrid(N/dimBlock.x, N/dimBlock.y);

    addMatrix<<<dimGrid,dimBlock>>>(pA,pB,pC,N);

    hipMemcpy(c, pC, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

	int i, j;
	/*
	printf("C = \n");
	for(i=0;i<N;i++){
	    for(j=0;j<N;j++){
	        printf("%d ", c[i][j]);
	    }
	    printf("\n");
	}
	*/

	hipFree(pA); 
	hipFree(pB); 
	hipFree(pC);

	printf("\n");

	return 0;
}
